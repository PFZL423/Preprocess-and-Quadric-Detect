#include "hip/hip_runtime.h"
#include "gpu_demo/QuadricDetect.h"
#include "gpu_demo/QuadricDetect_kernels.cuh"
#include <hipsolver.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <ctime>
#include <iostream>
#include <cmath>     // 添加这个头文件用于isfinite函数
#include <algorithm> // 添加这个头文件用于min函数

// ========================================
// CUDA内核函数定义 (每个内核只定义一次!)
// ========================================

__global__ void initCurandStates_Kernel(hiprandState *states, unsigned long seed, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void sampleAndBuildMatrices_Kernel(
    const GPUPoint3f *all_points,
    const int *remaining_indices,
    int num_remaining,
    hiprandState *rand_states,
    int batch_size,
    float *batch_matrices)
{
    int model_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (model_id >= batch_size)
        return;

    hiprandState local_state = rand_states[model_id];

    // 采样9个点
    int sample_indices[9];
    for (int i = 0; i < 9; ++i)
    {
        sample_indices[i] = remaining_indices[hiprand(&local_state) % num_remaining];
    }

    // 构造9x10的A矩阵 (🔧 修复：按列主序存储，符合cuSolver要求)
    float *A = &batch_matrices[model_id * 90]; // 9*10

    for (int i = 0; i < 9; ++i)
    {
        GPUPoint3f pt = all_points[sample_indices[i]];
        float x = pt.x, y = pt.y, z = pt.z;

        // 🔧 关键修复：检查并处理无效的点云数据
        if (!isfinite(x) || !isfinite(y) || !isfinite(z) ||
            isnan(x) || isnan(y) || isnan(z) ||
            isinf(x) || isinf(y) || isinf(z))
        {
            // 🚨 发现无效点，用默认值替换
            x = 0.0f;
            y = 0.0f;
            z = 0.0f;
        }

        // 🎯 关键修复：列主序存储 A[col * m + row]
        A[0 * 9 + i] = x * x; // x² (第0列)
        A[1 * 9 + i] = y * y; // y² (第1列)
        A[2 * 9 + i] = z * z; // z² (第2列)
        A[3 * 9 + i] = x * y; // xy (第3列)
        A[4 * 9 + i] = x * z; // xz (第4列)
        A[5 * 9 + i] = y * z; // yz (第5列)
        A[6 * 9 + i] = x;     // x  (第6列)
        A[7 * 9 + i] = y;     // y  (第7列)
        A[8 * 9 + i] = z;     // z  (第8列)
        A[9 * 9 + i] = 1.0f;  // 常数项 (第9列)

        // 🔧 二次验证：确保生成的值都是有效的
        for (int col = 0; col < 10; ++col)
        {
            float val = A[col * 9 + i];
            if (!isfinite(val) || isnan(val) || isinf(val))
            {
                A[col * 9 + i] = (col == 9) ? 1.0f : 0.0f; // 常数项设为1，其他设为0
            }
        }
    }

    rand_states[model_id] = local_state;
}

__global__ void countInliersBatch_Kernel(
    const GPUPoint3f *all_points,
    const int *remaining_indices,
    int num_remaining,
    const GPUQuadricModel *batch_models,
    int batch_size,
    float threshold,
    int *batch_inlier_counts)
{
    int model_id = blockIdx.y; // 使用2D grid，y维度对应模型
    if (model_id >= batch_size)
        return;

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int local_count = 0;

    // 每个线程处理多个点
    for (int i = thread_id; i < num_remaining; i += blockDim.x * gridDim.x)
    {
        GPUPoint3f point = all_points[remaining_indices[i]];
        float dist = evaluateQuadricDistance(point, batch_models[model_id]);

        if (dist < threshold)
        {
            local_count++;
        }
    }

    // Block内reduce求和
    __shared__ int shared_counts[256];
    shared_counts[threadIdx.x] = local_count;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            shared_counts[threadIdx.x] += shared_counts[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        atomicAdd(&batch_inlier_counts[model_id], shared_counts[0]);
    }
}

__device__ inline float evaluateQuadricDistance(
    const GPUPoint3f &point,
    const GPUQuadricModel &model)
{
    float x = point.x, y = point.y, z = point.z;

    // 🔧 修复开始：添加输入验证
    // 验证输入点的有效性
    if (!isfinite(x) || !isfinite(y) || !isfinite(z) ||
        isnan(x) || isnan(y) || isnan(z) ||
        isinf(x) || isinf(y) || isinf(z))
    {
        return 1e10f; // 返回一个很大的距离，表示无效点
    }

    // 验证模型系数的有效性
    bool model_valid = true;
    for (int i = 0; i < 16; ++i)
    {
        if (!isfinite(model.coeffs[i]) || isnan(model.coeffs[i]) || isinf(model.coeffs[i]))
        {
            model_valid = false;
            break;
        }
    }

    if (!model_valid)
    {
        return 1e10f; // 返回一个很大的距离，表示无效模型
    }
    // 🔧 修复结束

    // 手写二次型计算: [x y z 1] * Q * [x y z 1]^T
    float result = 0.0f;
    float coords[4] = {x, y, z, 1.0f};

    // 🔧 修复：使用更安全的矩阵乘法，避免潜在的内存访问问题
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            int idx = i * 4 + j;      // 确保索引在有效范围内
            if (idx >= 0 && idx < 16) // 🔧 添加边界检查
            {
                float coeff = model.coeffs[idx];
                // 🔧 验证每次乘法的结果
                float term = coords[i] * coeff * coords[j];
                if (isfinite(term) && !isnan(term) && !isinf(term))
                {
                    result += term;
                }
            }
        }
    }

    // 🔧 修复：验证最终结果的有效性
    if (!isfinite(result) || isnan(result) || isinf(result))
    {
        return 1e10f; // 返回一个很大的距离，表示计算失败
    }

    return fabsf(result);
}

__global__ void findBestModel_Kernel(
    const int *batch_inlier_counts,
    int batch_size,
    int *best_index,
    int *best_count)
{
    int thread_id = threadIdx.x;
    int local_best_idx = -1;
    int local_best_count = 0;

    // 每个线程处理多个模型
    for (int i = thread_id; i < batch_size; i += blockDim.x)
    {
        if (batch_inlier_counts[i] > local_best_count)
        {
            local_best_count = batch_inlier_counts[i];
            local_best_idx = i;
        }
    }

    // Block内reduce找最大值
    __shared__ int shared_counts[256];
    __shared__ int shared_indices[256];

    shared_counts[thread_id] = local_best_count;
    shared_indices[thread_id] = local_best_idx;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (thread_id < stride)
        {
            if (shared_counts[thread_id + stride] > shared_counts[thread_id])
            {
                shared_counts[thread_id] = shared_counts[thread_id + stride];
                shared_indices[thread_id] = shared_indices[thread_id + stride];
            }
        }
        __syncthreads();
    }

    if (thread_id == 0)
    {
        *best_count = shared_counts[0];
        *best_index = shared_indices[0];
    }
}

__global__ void extractInliers_Kernel(
    const GPUPoint3f *all_points,
    const int *remaining_indices,
    int num_remaining,
    const GPUQuadricModel *model,
    float threshold,
    int *inlier_indices,
    int *inlier_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_remaining)
        return;

    // 🔧 修复开始：添加更多安全检查
    // 检查输入参数有效性
    if (all_points == nullptr || remaining_indices == nullptr ||
        model == nullptr || inlier_indices == nullptr || inlier_count == nullptr)
    {
        return; // 静默返回，避免在GPU上打印错误
    }

    // 检查索引边界
    int global_point_index = remaining_indices[idx];
    if (global_point_index < 0)
    {
        return; // 无效的点索引
    }

    // 🔧 关键修复：确保我们不访问超出all_points数组边界的内存
    // 注意：我们无法在GPU内核中直接获取all_points的大小，所以需要依赖调用方确保索引有效

    GPUPoint3f point = all_points[global_point_index];

    // 🔧 验证点的有效性
    if (!isfinite(point.x) || !isfinite(point.y) || !isfinite(point.z) ||
        isnan(point.x) || isnan(point.y) || isnan(point.z) ||
        isinf(point.x) || isinf(point.y) || isinf(point.z))
    {
        return; // 跳过无效点
    }

    float dist = evaluateQuadricDistance(point, *model);

    // 🔧 验证距离计算结果的有效性
    if (!isfinite(dist) || isnan(dist) || isinf(dist))
    {
        return; // 跳过无效距离计算结果
    }
    // 🔧 修复结束

    if (dist < threshold)
    {
        // 🔧 修复开始：添加边界检查防止数组越界
        int write_pos = atomicAdd(inlier_count, 1);

        // 🔧 关键安全检查：确保不会越界访问
        // 理论上 d_temp_inlier_indices_ 大小等于 d_remaining_indices_.size()
        // 所以 write_pos 应该永远 < num_remaining，但为了安全还是检查
        if (write_pos < num_remaining)
        {
            inlier_indices[write_pos] = global_point_index;
        }
        else
        {
            // 🚨 如果发生越界，至少不会崩溃，但会丢失这个内点
            // 在实际应用中这种情况不应该发生
            atomicAdd(inlier_count, -1); // 回滚计数器
        }
        // 🔧 修复结束
    }
} // ========================================
// 成员函数实现 (每个函数只定义一次!)
// ========================================

void QuadricDetect::initializeGPUMemory(int batch_size)
{
    // 分配GPU内存
    d_batch_matrices_.resize(batch_size * 9 * 10);
    d_batch_models_.resize(batch_size);
    d_batch_inlier_counts_.resize(batch_size);
    d_rand_states_.resize(batch_size);

    // 初始化结果存储
    d_best_model_index_.resize(1);
    d_best_model_count_.resize(1);

    // 🆕 添加反幂迭代相关
    d_batch_ATA_matrices_.resize(batch_size * 10 * 10);
    d_batch_R_matrices_.resize(batch_size * 10 * 10);
    d_batch_eigenvectors_.resize(batch_size * 10);
}

void QuadricDetect::uploadPointsToGPU(const std::vector<GPUPoint3f> &h_points)
{
    d_all_points_ = h_points;
    d_remaining_indices_.resize(h_points.size());
    thrust::sequence(d_remaining_indices_.begin(), d_remaining_indices_.end(), 0);
}

void QuadricDetect::launchInitCurandStates(int batch_size)
{
    dim3 block(256);
    dim3 grid((batch_size + block.x - 1) / block.x);

    initCurandStates_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_rand_states_.data()),
        time(nullptr),
        batch_size);
    hipStreamSynchronize(stream_);
}

void QuadricDetect::launchSampleAndBuildMatrices(int batch_size)
{
    if (params_.verbosity > 0)
    {
        std::cout << "[launchSampleAndBuildMatrices] 开始生成批量矩阵，batch_size=" << batch_size << std::endl;
        std::cout << "  - 剩余点数: " << d_remaining_indices_.size() << std::endl;
        std::cout << "  - 总点数: " << d_all_points_.size() << std::endl;
    }

    // 🔍 验证输入数据
    if (d_remaining_indices_.size() < 9)
    {
        std::cerr << "[launchSampleAndBuildMatrices] 🚨 错误：剩余点数不足9个，无法生成矩阵！" << std::endl;
        return;
    }

    if (d_all_points_.size() == 0)
    {
        std::cerr << "[launchSampleAndBuildMatrices] 🚨 错误：点云数据为空！" << std::endl;
        return;
    }

    // 🔧 新增：验证点云数据的有效性
    if (params_.verbosity > 1)
    {
        std::cout << "[launchSampleAndBuildMatrices] 🔍 验证输入点云数据有效性..." << std::endl;

        // 检查前几个点的数据
        thrust::host_vector<GPUPoint3f> h_sample_points(std::min(10, (int)d_all_points_.size()));
        hipMemcpy(h_sample_points.data(),
                   thrust::raw_pointer_cast(d_all_points_.data()),
                   h_sample_points.size() * sizeof(GPUPoint3f),
                   hipMemcpyDeviceToHost);

        int invalid_points = 0;
        for (size_t i = 0; i < h_sample_points.size(); ++i)
        {
            const GPUPoint3f &pt = h_sample_points[i];
            if (!std::isfinite(pt.x) || !std::isfinite(pt.y) || !std::isfinite(pt.z) ||
                std::isnan(pt.x) || std::isnan(pt.y) || std::isnan(pt.z) ||
                std::isinf(pt.x) || std::isinf(pt.y) || std::isinf(pt.z))
            {
                invalid_points++;
                std::cout << "    🚨 发现无效点[" << i << "]: ("
                          << pt.x << ", " << pt.y << ", " << pt.z << ")" << std::endl;
            }
        }

        if (invalid_points > 0)
        {
            std::cout << "    🚨 警告：输入点云包含 " << invalid_points << " 个无效点！" << std::endl;
            std::cout << "    这可能导致SVD计算失败，建议预处理点云数据" << std::endl;
        }
        else
        {
            std::cout << "    ✓ 输入点云数据有效" << std::endl;
        }
    }

    dim3 block(256);
    dim3 grid((batch_size + block.x - 1) / block.x);

    // 🔍 先清零矩阵数据，确保没有垃圾数据
    thrust::fill(d_batch_matrices_.begin(), d_batch_matrices_.end(), 0.0f);

    sampleAndBuildMatrices_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_all_points_.data()),
        thrust::raw_pointer_cast(d_remaining_indices_.data()),
        static_cast<int>(d_remaining_indices_.size()),
        thrust::raw_pointer_cast(d_rand_states_.data()),
        batch_size,
        thrust::raw_pointer_cast(d_batch_matrices_.data()));

    hipError_t kernel_error = hipGetLastError();
    if (kernel_error != hipSuccess)
    {
        std::cerr << "[launchSampleAndBuildMatrices] 🚨 内核启动错误: " << hipGetErrorString(kernel_error) << std::endl;
        return;
    }

    hipStreamSynchronize(stream_);

    hipError_t sync_error = hipGetLastError();
    if (sync_error != hipSuccess)
    {
        std::cerr << "[launchSampleAndBuildMatrices] 🚨 内核执行错误: " << hipGetErrorString(sync_error) << std::endl;
        return;
    }

    // 🔍 验证生成的矩阵数据
    if (params_.verbosity > 1)
    {
        std::cout << "[launchSampleAndBuildMatrices] 验证生成的矩阵..." << std::endl;

        // 检查第一个矩阵
        thrust::host_vector<float> h_first_matrix(9 * 10);
        hipMemcpy(h_first_matrix.data(),
                   thrust::raw_pointer_cast(d_batch_matrices_.data()),
                   9 * 10 * sizeof(float),
                   hipMemcpyDeviceToHost);

        bool all_zero = true;
        for (int i = 0; i < 9 * 10; ++i)
        {
            if (h_first_matrix[i] != 0.0f)
            {
                all_zero = false;
                break;
            }
        }

        if (all_zero)
        {
            std::cerr << "[launchSampleAndBuildMatrices] 🚨 生成的矩阵全为零！检查内核实现" << std::endl;

            // 🔍 检查输入点云数据
            thrust::host_vector<GPUPoint3f> h_points_sample(std::min(10, (int)d_all_points_.size()));
            hipMemcpy(h_points_sample.data(),
                       thrust::raw_pointer_cast(d_all_points_.data()),
                       h_points_sample.size() * sizeof(GPUPoint3f),
                       hipMemcpyDeviceToHost);

            std::cout << "  - 前几个点云数据样本:" << std::endl;
            for (size_t i = 0; i < h_points_sample.size(); ++i)
            {
                std::cout << "    点" << i << ": (" << h_points_sample[i].x
                          << ", " << h_points_sample[i].y
                          << ", " << h_points_sample[i].z << ")" << std::endl;
            }

            // 🔍 检查剩余索引
            thrust::host_vector<int> h_indices_sample(std::min(10, (int)d_remaining_indices_.size()));
            hipMemcpy(h_indices_sample.data(),
                       thrust::raw_pointer_cast(d_remaining_indices_.data()),
                       h_indices_sample.size() * sizeof(int),
                       hipMemcpyDeviceToHost);

            std::cout << "  - 前几个剩余索引:" << std::endl;
            for (size_t i = 0; i < h_indices_sample.size(); ++i)
            {
                std::cout << "    索引" << i << ": " << h_indices_sample[i] << std::endl;
            }
        }
        else
        {
            std::cout << "[launchSampleAndBuildMatrices] ✓ 矩阵生成成功，包含非零数据" << std::endl;
        }
    }

    if (params_.verbosity > 0)
    {
        std::cout << "[launchSampleAndBuildMatrices] 矩阵生成完成" << std::endl;
    }
}

void QuadricDetect::launchCountInliersBatch(int batch_size)
{
    // 修复: 使用2D grid匹配内核实现
    dim3 block(256);
    dim3 grid_x((d_remaining_indices_.size() + block.x - 1) / block.x);
    dim3 grid(grid_x.x, batch_size); // 2D grid: (points, models)

    // 先清零计数器
    thrust::fill(d_batch_inlier_counts_.begin(), d_batch_inlier_counts_.end(), 0);

    countInliersBatch_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_all_points_.data()),
        thrust::raw_pointer_cast(d_remaining_indices_.data()),
        static_cast<int>(d_remaining_indices_.size()),
        thrust::raw_pointer_cast(d_batch_models_.data()),
        batch_size,
        static_cast<float>(params_.quadric_distance_threshold),
        thrust::raw_pointer_cast(d_batch_inlier_counts_.data()));
    hipStreamSynchronize(stream_);
}

void QuadricDetect::launchFindBestModel(int batch_size)
{
    findBestModel_Kernel<<<1, 256, 0, stream_>>>(
        thrust::raw_pointer_cast(d_batch_inlier_counts_.data()),
        batch_size,
        thrust::raw_pointer_cast(d_best_model_index_.data()),
        thrust::raw_pointer_cast(d_best_model_count_.data()));
    hipStreamSynchronize(stream_);
}

// 替换你 QuadricDetect.cu 文件中的占位符实现：
void QuadricDetect::launchExtractInliers(const GPUQuadricModel *model)
{
    if (params_.verbosity > 0)
    {
        std::cout << "[launchExtractInliers] 开始提取内点索引" << std::endl;
    }

    // 🔧 修复开始：添加详细的输入验证
    std::cout << "debug1" << std::endl;

    // 验证输入参数
    if (model == nullptr)
    {
        std::cerr << "[launchExtractInliers] 🚨 错误：model指针为空！" << std::endl;
        current_inlier_count_ = 0;
        return;
    }

    if (d_remaining_indices_.size() == 0)
    {
        std::cerr << "[launchExtractInliers] 🚨 错误：没有剩余点可处理！" << std::endl;
        current_inlier_count_ = 0;
        return;
    }

    if (d_all_points_.size() == 0)
    {
        std::cerr << "[launchExtractInliers] 🚨 错误：点云数据为空！" << std::endl;
        current_inlier_count_ = 0;
        return;
    }

    std::cout << "  - 剩余点数: " << d_remaining_indices_.size() << std::endl;
    std::cout << "  - 总点数: " << d_all_points_.size() << std::endl;
    std::cout << "  - 距离阈值: " << params_.quadric_distance_threshold << std::endl;

    // 🔧 关键修复：将model从CPU拷贝到GPU专用内存
    thrust::device_vector<GPUQuadricModel> d_model_safe(1);
    d_model_safe[0] = *model; // 安全拷贝
    std::cout << "debug1.5 - 模型已安全拷贝到GPU" << std::endl;
    // 🔧 修复结束

    // 分配临时GPU内存存储内点索引
    d_temp_inlier_indices_.resize(d_remaining_indices_.size());
    std::cout << "debug2" << std::endl;
    thrust::device_vector<int> d_inlier_count(1, 0);
    std::cout << "debug3" << std::endl;

    // 配置CUDA网格
    dim3 block(256);
    dim3 grid((d_remaining_indices_.size() + block.x - 1) / block.x);
    std::cout << "debug3.5 - Grid配置: " << grid.x << " blocks, " << block.x << " threads" << std::endl;

    // 🔧 修复：使用安全的GPU内存而不是CPU指针
    extractInliers_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_all_points_.data()),
        thrust::raw_pointer_cast(d_remaining_indices_.data()),
        static_cast<int>(d_remaining_indices_.size()),
        thrust::raw_pointer_cast(d_model_safe.data()), // 🔧 使用GPU内存
        static_cast<float>(params_.quadric_distance_threshold),
        thrust::raw_pointer_cast(d_temp_inlier_indices_.data()),
        thrust::raw_pointer_cast(d_inlier_count.data()));
    std::cout << "debug4" << std::endl;

    hipStreamSynchronize(stream_);
    std::cout << "debug5" << std::endl;

    // 🔧 修复开始：使用更安全的内存访问方法替代thrust::copy
    // 检查内核执行是否有错误
    hipError_t kernel_error = hipGetLastError();
    if (kernel_error != hipSuccess)
    {
        std::cerr << "[launchExtractInliers] 🚨 内核执行错误: " << hipGetErrorString(kernel_error) << std::endl;
        current_inlier_count_ = 0;
        return;
    }

    // 获取内点数量并调整大小
    // 🚨 原始代码 - 可能导致非法内存访问：
    // thrust::host_vector<int> h_count = d_inlier_count;
    // thrust::host_vector<int> h_count(1);
    // thrust::copy(d_inlier_count.begin(), d_inlier_count.end(), h_count.begin());

    // 🔧 新方案：使用原生hipMemcpy，更安全可控
    int h_count_temp = 0;
    hipError_t copy_error = hipMemcpy(&h_count_temp,
                                        thrust::raw_pointer_cast(d_inlier_count.data()),
                                        sizeof(int),
                                        hipMemcpyDeviceToHost);

    if (copy_error != hipSuccess)
    {
        std::cerr << "[launchExtractInliers] 🚨 内存拷贝错误: " << hipGetErrorString(copy_error) << std::endl;
        current_inlier_count_ = 0;
        return;
    }

    current_inlier_count_ = h_count_temp;
    // 🔧 修复结束

    std::cout << "debug6" << std::endl;

    // 🚨 原始代码已移除 - 会导致编译错误：
    // current_inlier_count_ = h_count[0];

    std::cout << "debug7" << std::endl;

    d_temp_inlier_indices_.resize(current_inlier_count_);
    std::cout << "debug8" << std::endl;

    if (params_.verbosity > 0)
    {
        std::cout << "[launchExtractInliers] 找到 " << current_inlier_count_ << " 个内点" << std::endl;
    }
}

void QuadricDetect::getBestModelResults(thrust::host_vector<int> &h_best_index, thrust::host_vector<int> &h_best_count)
{
    // 从device拷贝到host
    h_best_index = d_best_model_index_;
    h_best_count = d_best_model_count_;
}

// remove的GPU函数实现
// 在 QuadricDetect.cu 中添加内核
__global__ void removePointsKernel(
    const int *remaining_points,
    int remaining_count,
    const int *sorted_inliers, // 已排序的内点索引
    int inlier_count,
    int *output_points,
    int *output_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= remaining_count)
        return;

    int point_id = remaining_points[idx];

    // GPU上二分查找
    bool is_inlier = false;
    int left = 0, right = inlier_count - 1;
    while (left <= right)
    {
        int mid = (left + right) / 2;
        if (sorted_inliers[mid] == point_id)
        {
            is_inlier = true;
            break;
        }
        if (sorted_inliers[mid] < point_id)
            left = mid + 1;
        else
            right = mid - 1;
    }

    // 如果不是内点，就保留
    if (!is_inlier)
    {
        int write_pos = atomicAdd(output_count, 1);
        output_points[write_pos] = point_id;
    }
}

// 包装函数
void QuadricDetect::launchRemovePointsKernel()
{
    // 1. 对内点索引排序（纯GPU操作）
    thrust::sort(d_temp_inlier_indices_.begin(),
                 d_temp_inlier_indices_.begin() + current_inlier_count_);

    // 2. 分配输出空间
    thrust::device_vector<int> d_new_remaining(d_remaining_indices_.size());
    thrust::device_vector<int> d_output_count(1, 0);

    // 3. 启动内核
    dim3 block(256);
    dim3 grid((d_remaining_indices_.size() + block.x - 1) / block.x);

    removePointsKernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_remaining_indices_.data()),
        static_cast<int>(d_remaining_indices_.size()),
        thrust::raw_pointer_cast(d_temp_inlier_indices_.data()),
        current_inlier_count_,
        thrust::raw_pointer_cast(d_new_remaining.data()),
        thrust::raw_pointer_cast(d_output_count.data()));

    hipStreamSynchronize(stream_);

    // 4. 获取实际输出大小并调整
    thrust::host_vector<int> h_count = d_output_count;
    int new_size = h_count[0]; // 🚨 这里有一次小传输，但unavoidable

    d_new_remaining.resize(new_size);
    d_remaining_indices_ = std::move(d_new_remaining);
}

// 🆕 新增函数实现--反幂迭代的核心实现
// 🆕 添加到QuadricDetect.cu

// 1. 计算A^T*A矩阵
__global__ void computeATA_Kernel(
    const float *batch_matrices, // 输入：1024个9×10矩阵
    float *batch_ATA_matrices,   // 输出：1024个10×10 A^T*A矩阵
    int batch_size)
{
    int batch_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_id >= batch_size)
        return;

    const float *A = &batch_matrices[batch_id * 90];  // 9×10矩阵
    float *ATA = &batch_ATA_matrices[batch_id * 100]; // 10×10矩阵

    // 计算A^T * A
    for (int i = 0; i < 10; ++i)
    {
        for (int j = i; j < 10; ++j)
        { // 只计算上三角，利用对称性
            float sum = 0.0f;
            for (int k = 0; k < 9; ++k)
            {
                sum += A[i * 9 + k] * A[j * 9 + k]; // A^T[i][k] * A[j][k]
            }
            ATA[i * 10 + j] = sum;
            ATA[j * 10 + i] = sum; // 对称矩阵
        }
    }
}

__global__ void batchQR_Kernel(
    const float *batch_ATA_matrices,
    float *batch_R_matrices,
    int batch_size)
{
    int batch_id = blockIdx.x;
    if (batch_id >= batch_size)
        return;

    __shared__ float A[10][10];
    __shared__ float R[10][10];

    // 🔧 1. 先初始化R矩阵为零
    for (int i = threadIdx.x; i < 100; i += blockDim.x)
    {
        ((float *)R)[i] = 0.0f;
    }
    __syncthreads();

    // 🔧 2. 加载A^T*A到共享内存
    const float *ATA = &batch_ATA_matrices[batch_id * 100];
    for (int i = threadIdx.x; i < 100; i += blockDim.x)
    {
        ((float *)A)[i] = ATA[i];
    }
    __syncthreads();

    // 🔧 3. 执行Gram-Schmidt QR分解
    for (int k = 0; k < 10; ++k)
    {
        if (threadIdx.x == 0)
        {
            // 计算第k列的模长
            float norm_sq = 0.0f;
            for (int i = k; i < 10; ++i)
            {
                norm_sq += A[i][k] * A[i][k];
            }
            float norm = sqrtf(norm_sq);

            // 数值稳定性检查
            if (norm < 1e-12f)
            {
                for (int i = k; i < 10; ++i)
                {
                    A[i][k] = (i == k) ? 1.0f : 0.0f;
                }
                norm = 1.0f;
            }

            // 归一化第k列
            for (int i = k; i < 10; ++i)
            {
                A[i][k] /= norm;
            }

            // 🎯 设置R[k][k] (对角线元素)
            R[k][k] = norm;

            // 正交化后续列
            for (int j = k + 1; j < 10; ++j)
            {
                // 计算投影系数
                float proj_coeff = 0.0f;
                for (int i = k; i < 10; ++i)
                {
                    proj_coeff += A[i][k] * A[i][j];
                }

                // 🎯 设置R[k][j] (上三角元素)
                R[k][j] = proj_coeff;

                // 从a_j中减去投影
                for (int i = k; i < 10; ++i)
                {
                    A[i][j] -= proj_coeff * A[i][k];
                }
            }
        }
        __syncthreads();
    }

    // 🔧 4. 输出R矩阵 (不要再清零了!)
    float *R_out = &batch_R_matrices[batch_id * 100];
    for (int i = threadIdx.x; i < 100; i += blockDim.x)
    {
        R_out[i] = ((float *)R)[i];
    }
}

// 3. 反幂迭代内核
__global__ void batchInversePowerIteration_Kernel(
    const float *batch_R_matrices, // 输入：1024个10×10 R矩阵
    float *batch_eigenvectors,     // 输出：1024个10维最小特征向量
    hiprandState *rand_states,      // 随机数状态
    int batch_size)
{
    int batch_id = blockIdx.x;
    if (batch_id >= batch_size)
        return;

    __shared__ float R[10][10]; // R矩阵
    __shared__ float x[10];     // 当前向量
    __shared__ float y[10];     // 临时向量

    // 加载R矩阵
    const float *R_in = &batch_R_matrices[batch_id * 100];
    for (int i = threadIdx.x; i < 100; i += blockDim.x)
    {
        ((float *)R)[i] = R_in[i];
    }

    // 初始化随机向量
    if (threadIdx.x < 10)
    {
        hiprandState local_state = rand_states[batch_id * 10 + threadIdx.x];
        x[threadIdx.x] = hiprand_uniform(&local_state);
        rand_states[batch_id * 10 + threadIdx.x] = local_state;
    }
    __syncthreads();

    // 反幂迭代：8次迭代
    for (int iter = 0; iter < 8; ++iter)
    {
        // 解 R * y = x (回代法)
        if (threadIdx.x == 0)
        {
            for (int i = 9; i >= 0; --i)
            {
                float sum = x[i];
                for (int j = i + 1; j < 10; ++j)
                {
                    sum -= R[i][j] * y[j];
                }
                y[i] = (fabsf(R[i][i]) > 1e-12f) ? sum / R[i][i] : 0.0f;
            }
        }
        __syncthreads();

        // 归一化 y -> x
        if (threadIdx.x == 0)
        {
            float norm = 0.0f;
            for (int i = 0; i < 10; ++i)
            {
                norm += y[i] * y[i];
            }
            norm = sqrtf(norm);
            if (norm > 1e-12f)
            {
                for (int i = 0; i < 10; ++i)
                {
                    x[i] = y[i] / norm;
                }
            }
        }
        __syncthreads();
    }

    // 输出最终特征向量
    float *output = &batch_eigenvectors[batch_id * 10];
    if (threadIdx.x < 10)
    {
        output[threadIdx.x] = x[threadIdx.x];
    }
}

// 4. 提取二次曲面模型内核
__global__ void extractQuadricModels_Kernel(
    const float *batch_eigenvectors, // 输入：1024个10维特征向量
    GPUQuadricModel *batch_models,   // 输出：1024个二次曲面模型
    int batch_size)
{
    int batch_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_id >= batch_size)
        return;

    const float *eigenvec = &batch_eigenvectors[batch_id * 10];
    GPUQuadricModel *model = &batch_models[batch_id];

    // 初始化coeffs数组
    for (int i = 0; i < 16; ++i)
    {
        model->coeffs[i] = 0.0f;
    }

    // L2归一化
    float norm_sq = 0.0f;
    for (int i = 0; i < 10; ++i)
    {
        norm_sq += eigenvec[i] * eigenvec[i];
    }
    float norm_factor = (norm_sq > 1e-12f) ? 1.0f / sqrtf(norm_sq) : 1.0f;

    // 10维向量→16维coeffs的映射 (4x4对称矩阵按行主序存储)
    // 二次曲面方程: Ax² + By² + Cz² + 2Dxy + 2Exz + 2Fyz + 2Gx + 2Hy + 2Iz + J = 0
    // 对应特征向量: [A, B, C, D, E, F, G, H, I, J]

    float A = eigenvec[0] * norm_factor; // x²系数
    float B = eigenvec[1] * norm_factor; // y²系数
    float C = eigenvec[2] * norm_factor; // z²系数
    float D = eigenvec[3] * norm_factor; // xy系数
    float E = eigenvec[4] * norm_factor; // xz系数
    float F = eigenvec[5] * norm_factor; // yz系数
    float G = eigenvec[6] * norm_factor; // x系数
    float H = eigenvec[7] * norm_factor; // y系数
    float I = eigenvec[8] * norm_factor; // z系数
    float J = eigenvec[9] * norm_factor; // 常数项

    // 4×4对称矩阵Q的映射 (按行主序存储到coeffs[16])
    // Q = [[A,   D,   E,   G],
    //      [D,   B,   F,   H],
    //      [E,   F,   C,   I],
    //      [G,   H,   I,   J]]

    model->coeffs[0] = A;  // Q(0,0)
    model->coeffs[1] = D;  // Q(0,1)
    model->coeffs[2] = E;  // Q(0,2)
    model->coeffs[3] = G;  // Q(0,3)
    model->coeffs[4] = D;  // Q(1,0) = Q(0,1)
    model->coeffs[5] = B;  // Q(1,1)
    model->coeffs[6] = F;  // Q(1,2)
    model->coeffs[7] = H;  // Q(1,3)
    model->coeffs[8] = E;  // Q(2,0) = Q(0,2)
    model->coeffs[9] = F;  // Q(2,1) = Q(1,2)
    model->coeffs[10] = C; // Q(2,2)
    model->coeffs[11] = I; // Q(2,3)
    model->coeffs[12] = G; // Q(3,0) = Q(0,3)
    model->coeffs[13] = H; // Q(3,1) = Q(1,3)
    model->coeffs[14] = I; // Q(3,2) = Q(2,3)
    model->coeffs[15] = J; // Q(3,3)
}

// 包装函数
// 🆕 添加到QuadricDetect.cu

void QuadricDetect::launchComputeATA(int batch_size)
{
    dim3 block(256);
    dim3 grid((batch_size + block.x - 1) / block.x);

    computeATA_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_batch_matrices_.data()),
        thrust::raw_pointer_cast(d_batch_ATA_matrices_.data()),
        batch_size);
    hipStreamSynchronize(stream_);
}

void QuadricDetect::launchBatchQR(int batch_size)
{
    dim3 block(256);
    dim3 grid(batch_size); // 每个block处理一个矩阵

    batchQR_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_batch_ATA_matrices_.data()),
        thrust::raw_pointer_cast(d_batch_R_matrices_.data()),
        batch_size);
    hipStreamSynchronize(stream_);
}

void QuadricDetect::launchBatchInversePower(int batch_size)
{
    dim3 block(256);
    dim3 grid(batch_size); // 每个block处理一个矩阵

    batchInversePowerIteration_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_batch_R_matrices_.data()),
        thrust::raw_pointer_cast(d_batch_eigenvectors_.data()),
        thrust::raw_pointer_cast(d_rand_states_.data()),
        batch_size);
    hipStreamSynchronize(stream_);
}

void QuadricDetect::launchExtractQuadricModels(int batch_size)
{
    dim3 block(256);
    dim3 grid((batch_size + block.x - 1) / block.x);

    extractQuadricModels_Kernel<<<grid, block, 0, stream_>>>(
        thrust::raw_pointer_cast(d_batch_eigenvectors_.data()),
        thrust::raw_pointer_cast(d_batch_models_.data()),
        batch_size);
    hipStreamSynchronize(stream_);
}

// 重载实现
void QuadricDetect::uploadPointsToGPU(const thrust::device_vector<GPUPoint3f> &h_points)
{
    d_all_points_ = h_points;
    d_remaining_indices_.resize(h_points.size());
    thrust::sequence(d_remaining_indices_.begin(), d_remaining_indices_.end(), 0);
}
